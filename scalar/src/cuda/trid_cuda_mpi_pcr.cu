#include "hip/hip_runtime.h"
#include "trid_mpi.h"
#include "trid_strided_multidim_pcr_mpi.hpp"
#include "cutil_inline.h"

#include <cmath>

void setStartEnd(int *start, int *end, int coord, int numProcs, int numElements) {
  int tmp = numElements / numProcs;
  int remainder = numElements % numProcs;
  int total = 0;
  for(int i = 0; i < coord; i++) {
    if(i < remainder) {
      total += tmp + 1;
    } else {
      total += tmp;
    }
  }
  *start = total;
  if(coord < remainder) {
    *end = *start + tmp;
  } else {
    *end = *start + tmp -1;
  }
}

void rms(char* name, FP* array, trid_handle<FP> &handle, trid_mpi_handle &mpi_handle) {
  //Sum the square of values in app.h_u
  double sum = 0.0;
  for(int k = 0; k < handle.size[2]; k++) {
    for(int j = 0; j < handle.size[1]; j++) {
      for(int i = 0; i < handle.size[0]; i++) {
        int ind = k * handle.pads[0] * handle.pads[1] + j * handle.pads[0] + i;
        //sum += array[ind]*array[ind];
        sum += array[ind];
      }
    }
  }

  double global_sum = 0.0;
  MPI_Allreduce(&sum, &global_sum,1, MPI_DOUBLE,MPI_SUM, mpi_handle.comm);

  if(mpi_handle.rank ==0) {
    printf("%s sum = %lg\n", name, global_sum);
    //printf("%s rms = %2.15lg\n",name, sqrt(global_sum)/((double)(app.nx_g*app.ny_g*app.nz_g)));
  }

}

template<typename REAL>
void tridMultiDimBatchPCRInitMPI(trid_handle<REAL> &handle, trid_mpi_handle &mpi_handle, 
                                 int ndim, int *size) {
  // Get number of mpi procs and the rank of this mpi proc
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_handle.procs);
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_handle.rank);
  
  // Split into multi dim arrangement of mpi procs
  handle.ndim = ndim;
  mpi_handle.pdims    = (int *) calloc(handle.ndim, sizeof(int));
  mpi_handle.periodic = (int *) calloc(handle.ndim, sizeof(int)); //false
  mpi_handle.coords   = (int *) calloc(handle.ndim, sizeof(int));
  MPI_Dims_create(mpi_handle.procs, handle.ndim, mpi_handle.pdims);
  
  // Create cartecian mpi comm
  MPI_Cart_create(MPI_COMM_WORLD, handle.ndim, mpi_handle.pdims, mpi_handle.periodic, 0,  &mpi_handle.comm);
  
  // Get rand and coord of current mpi proc
  MPI_Comm_rank(mpi_handle.comm, &mpi_handle.my_cart_rank);
  MPI_Cart_coords(mpi_handle.comm, mpi_handle.my_cart_rank, handle.ndim, mpi_handle.coords);
  
  // TODO extend to other dimensions
  // Create separate comms for x, y and z dimensions
  int free_coords[3];
  free_coords[0] = 1;
  free_coords[1] = 0;
  free_coords[2] = 0;
  MPI_Cart_sub(mpi_handle.comm, free_coords, &mpi_handle.x_comm);
  MPI_Comm y_comm;
  free_coords[0] = 0;
  free_coords[1] = 1;
  free_coords[2] = 0;
  MPI_Cart_sub(mpi_handle.comm, free_coords, &mpi_handle.y_comm);
  MPI_Comm z_comm;
  free_coords[0] = 0;
  free_coords[1] = 0;
  free_coords[2] = 1;
  MPI_Cart_sub(mpi_handle.comm, free_coords, &mpi_handle.z_comm);
  
  // Store the global problem sizes
  handle.size_g = (int *) calloc(handle.ndim, sizeof(int));
  for(int i = 0; i < handle.ndim; i++) {
    handle.size_g[i] = size[i];
  }
  
  // Calculate size, padding, start and end for each dimension
  handle.size    = (int *) calloc(handle.ndim, sizeof(int));
  handle.pads    = (int *) calloc(handle.ndim, sizeof(int));
  handle.start_g = (int *) calloc(handle.ndim, sizeof(int));
  handle.end_g   = (int *) calloc(handle.ndim, sizeof(int));
  
  for(int i = 0; i < handle.ndim; i++) {
    setStartEnd(&handle.start_g[i], &handle.end_g[i], mpi_handle.coords[i], mpi_handle.pdims[i], 
                handle.size_g[i]);
    
    handle.size[i]    = handle.end_g[i] - handle.start_g[i] + 1;
    
    // Only pad the x dimension
    if(i == 0) {
      // TODO see what padding is needed for GPU
      //handle.pads[i] = (1 + ((handle.size[i] - 1) / SIMD_VEC)) * SIMD_VEC;
      handle.pads[i] = handle.size[i];
    } else {
      handle.pads[i] = handle.size[i];
    }
  }
  
  // Allocate memory for arrays
  int mem_size = sizeof(REAL);
  for(int i = 0; i < handle.ndim; i++) {
    mem_size *= handle.pads[i];
  }
  
  cudaSafeCall( hipMalloc((void **)&handle.a, mem_size) );
  cudaSafeCall( hipMalloc((void **)&handle.b, mem_size) );
  cudaSafeCall( hipMalloc((void **)&handle.c, mem_size) );
  cudaSafeCall( hipMalloc((void **)&handle.du, mem_size) );
  cudaSafeCall( hipMalloc((void **)&handle.h_u, mem_size) );
  
  // Calculate reduced system sizes for each dimension
  handle.sys_len_l = (int *) calloc(handle.ndim, sizeof(int));
  handle.n_sys_g = (int *) calloc(handle.ndim, sizeof(int));
  handle.n_sys_l = (int *) calloc(handle.ndim, sizeof(int));
  
  for(int i = 0; i < handle.ndim; i++) {
    handle.sys_len_l[i] = mpi_handle.pdims[i] * 2;
    handle.n_sys_g[i] = 1;
    handle.n_sys_l[i] = 1;
    for(int j = 0; j < handle.ndim; j++) {
      if(j != i) {
        handle.n_sys_g[i] *= handle.size[j];
        handle.n_sys_l[i] *= handle.size[j];
      }
    }
  }
}

template<typename REAL>
void tridMultiDimBatchPCRCleanMPI(trid_handle<REAL> &handle, trid_mpi_handle &mpi_handle) {
  free(mpi_handle.pdims);
  free(mpi_handle.periodic);
  free(mpi_handle.coords);
  free(handle.size_g);
  free(handle.size);
  free(handle.start_g);
  free(handle.end_g);
  free(handle.sys_len_l);
  free(handle.n_sys_g);
  free(handle.n_sys_l);
  cudaSafeCall( hipFree(handle.a) );
  cudaSafeCall( hipFree(handle.b) );
  cudaSafeCall( hipFree(handle.c) );
  cudaSafeCall( hipFree(handle.du) );
  cudaSafeCall( hipFree(handle.h_u) );
}

template<typename REAL, int INC>
void tridMultiDimBatchPCRSolveMPI(trid_handle<REAL> &handle, trid_mpi_handle &mpi_handle, 
                                  int solvedim) {
  // For now assume 1 MPI proc per GPU
  
  // Allocate aa, cc, dd
  REAL *aa = NULL;
  REAL *cc = NULL;
  REAL *dd = NULL;
  cudaSafeCall( hipMalloc((void **)&aa, sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2]) );
  cudaSafeCall( hipMalloc((void **)&cc, sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2]) );
  cudaSafeCall( hipMalloc((void **)&dd, sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2]) );
  
  if(solvedim == 0) {
    // Call forwards pass
    const int numTrids = handle.size[1] * handle.size[2];
    const int length = handle.size[0];
    const int stride = 1;
    const int batchSize = 1;
    const int batchStride = handle.pads[0];
    const int regStoreSize = 8;
    const int threadsPerTrid = (int)ceil((double)length / (double)regStoreSize);
    
    // Work out number of blocks and threads needed
    int totalThreads = threadsPerTrid * numTrids;
    int nThreads = 512;
    int nBlocks = 1;
    if(totalThreads < 512) {
      nThreads = totalThreads;
    } else {
      nBlocks = (int)ceil((double)totalThreads / (double)nThreads);
    }
    
    int reducedSize = threadsPerTrid * 2;
    // TODO change to one interwoven array once algorithm is working
    REAL *aa_r = NULL;
    REAL *cc_r = NULL;
    REAL *dd_r = NULL;
    cudaSafeCall( hipMalloc((void **)&aa_r, sizeof(REAL) * reducedSize * numTrids) );
    cudaSafeCall( hipMalloc((void **)&cc_r, sizeof(REAL) * reducedSize * numTrids) );
    cudaSafeCall( hipMalloc((void **)&dd_r, sizeof(REAL) * reducedSize * numTrids) );
    
    // Call forwards pass
    batched_trid_forwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(handle.a, handle.b, handle.c, 
                                handle.du, aa, cc, dd, aa_r, cc_r, dd_r, length, stride, 
                                numTrids, batchSize, batchStride, threadsPerTrid);
    // Check for errors in kernel
    cudaSafeCall( hipPeekAtLastError() );
    cudaSafeCall( hipDeviceSynchronize() );
    
    REAL *h_aa = (REAL *) malloc(sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2]);
    REAL *h_cc = (REAL *) malloc(sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2]);
    REAL *h_dd = (REAL *) malloc(sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2]);
    
    cudaSafeCall( hipMemcpy(h_aa, aa, sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2], hipMemcpyDeviceToHost) );
    cudaSafeCall( hipMemcpy(h_cc, cc, sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2], hipMemcpyDeviceToHost) );
    cudaSafeCall( hipMemcpy(h_dd, dd, sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2], hipMemcpyDeviceToHost) );
    
    rms("aa", h_aa, handle, mpi_handle);
    rms("cc", h_cc, handle, mpi_handle);
    rms("dd", h_dd, handle, mpi_handle);
    
    free(h_aa);
    free(h_cc);
    free(h_dd);
    
    // Call PCR reduced (modified to include MPI comm as reduced system will 
    // be spread over nodes)
    batched_trid_reduced<REAL, regStoreSize>(aa_r, cc_r, dd_r, numTrids, reducedSize, solvedim, threadsPerTrid, 
                               nBlocks, nThreads, length, mpi_handle);
    
    // Call backwards pass
    if(INC) {
      batched_trid_backwardsInc_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.h_u, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
      // Check for errors in kernel
      cudaSafeCall( hipPeekAtLastError() );
      cudaSafeCall( hipDeviceSynchronize() );
    } else {
      batched_trid_backwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.du, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
      // Check for errors in kernel
      cudaSafeCall( hipPeekAtLastError() );
      cudaSafeCall( hipDeviceSynchronize() );
    }
    
    // Free memory
    cudaSafeCall( hipFree(aa_r) );
    cudaSafeCall( hipFree(cc_r) );
    cudaSafeCall( hipFree(dd_r) );
  } else if(solvedim == 1) {
    // Call forwards pass
    const int numTrids = handle.size[0] * handle.size[2];
    const int length = handle.size[1];
    const int stride = handle.pads[0];
    const int batchSize = handle.size[0];
    const int batchStride = handle.pads[0] * handle.size[1];
    const int regStoreSize = 8;
    const int threadsPerTrid = (int)ceil((double)length / (double)regStoreSize);
    
    // Work out number of blocks and threads needed
    int totalThreads = threadsPerTrid * numTrids;
    int nThreads = 512;
    int nBlocks = 1;
    if(totalThreads < 512) {
      nThreads = totalThreads;
    } else {
      nBlocks = (int)ceil((double)totalThreads / (double)nThreads);
    }
    
    int reducedSize = threadsPerTrid * 2;
    // TODO change to one interwoven array once algorithm is working
    REAL *aa_r = NULL;
    REAL *cc_r = NULL;
    REAL *dd_r = NULL;
    cudaSafeCall( hipMalloc((void **)&aa_r, sizeof(REAL) * reducedSize * numTrids) );
    cudaSafeCall( hipMalloc((void **)&cc_r, sizeof(REAL) * reducedSize * numTrids) );
    cudaSafeCall( hipMalloc((void **)&dd_r, sizeof(REAL) * reducedSize * numTrids) );
    
    // Call forwards pass
    batched_trid_forwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(handle.a, handle.b, handle.c, 
                                handle.du, aa, cc, dd, aa_r, cc_r, dd_r, length, stride, 
                                numTrids, batchSize, batchStride, threadsPerTrid);
    // Check for errors in kernel
    cudaSafeCall( hipPeekAtLastError() );
    cudaSafeCall( hipDeviceSynchronize() );
    
    // Call PCR reduced (modified to include MPI comm as reduced system will 
    // be spread over nodes)
    batched_trid_reduced<REAL, regStoreSize>(aa_r, cc_r, dd_r, numTrids, reducedSize, solvedim, threadsPerTrid, 
                               nBlocks, nThreads, length, mpi_handle);
    
    // Call backwards pass
    if(INC) {
      batched_trid_backwardsInc_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.h_u, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
      // Check for errors in kernel
      cudaSafeCall( hipPeekAtLastError() );
      cudaSafeCall( hipDeviceSynchronize() );
    } else {
      batched_trid_backwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.du, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
      // Check for errors in kernel
      cudaSafeCall( hipPeekAtLastError() );
      cudaSafeCall( hipDeviceSynchronize() );
    }
    
    // Free memory
    cudaSafeCall( hipFree(aa_r) );
    cudaSafeCall( hipFree(cc_r) );
    cudaSafeCall( hipFree(dd_r) );
  } else if(solvedim == 2) {
    // Call forwards pass
    const int numTrids = handle.size[0] * handle.size[1];
    const int length = handle.size[2];
    const int stride = handle.pads[0] * handle.size[1];
    const int batchSize = handle.size[0];
    const int batchStride = handle.pads[0];
    const int regStoreSize = 8;
    const int threadsPerTrid = (int)ceil((double)length / (double)regStoreSize);
    
    // Work out number of blocks and threads needed
    int totalThreads = threadsPerTrid * numTrids;
    int nThreads = 512;
    int nBlocks = 1;
    if(totalThreads < 512) {
      nThreads = totalThreads;
    } else {
      nBlocks = (int)ceil((double)totalThreads / (double)nThreads);
    }
    
    int reducedSize = threadsPerTrid * 2;
    // TODO change to one interwoven array once algorithm is working
    REAL *aa_r = NULL;
    REAL *cc_r = NULL;
    REAL *dd_r = NULL;
    cudaSafeCall( hipMalloc((void **)&aa_r, sizeof(REAL) * reducedSize * numTrids) );
    cudaSafeCall( hipMalloc((void **)&cc_r, sizeof(REAL) * reducedSize * numTrids) );
    cudaSafeCall( hipMalloc((void **)&dd_r, sizeof(REAL) * reducedSize * numTrids) );
    
    // Call forwards pass
    batched_trid_forwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(handle.a, handle.b, handle.c, 
                                handle.du, aa, cc, dd, aa_r, cc_r, dd_r, length, stride, 
                                numTrids, batchSize, batchStride, threadsPerTrid);
    // Check for errors in kernel
    cudaSafeCall( hipPeekAtLastError() );
    cudaSafeCall( hipDeviceSynchronize() );
    
    // Call PCR reduced (modified to include MPI comm as reduced system will 
    // be spread over nodes)
    batched_trid_reduced<REAL, regStoreSize>(aa_r, cc_r, dd_r, numTrids, reducedSize, solvedim, threadsPerTrid, 
                               nBlocks, nThreads, length, mpi_handle);
    
    // Call backwards pass
    if(INC) {
      batched_trid_backwardsInc_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.h_u, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
      // Check for errors in kernel
      cudaSafeCall( hipPeekAtLastError() );
      cudaSafeCall( hipDeviceSynchronize() );
    } else {
      batched_trid_backwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.du, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
      // Check for errors in kernel
      cudaSafeCall( hipPeekAtLastError() );
      cudaSafeCall( hipDeviceSynchronize() );
    }
    
    // Free memory
    cudaSafeCall( hipFree(aa_r) );
    cudaSafeCall( hipFree(cc_r) );
    cudaSafeCall( hipFree(dd_r) );
  }
}

template void tridMultiDimBatchPCRInitMPI<float>(trid_handle<float> &handle, 
                                              trid_mpi_handle &mpi_handle, int ndim, int *size);

template void tridMultiDimBatchPCRInitMPI<double>(trid_handle<double> &handle, 
                                              trid_mpi_handle &mpi_handle, int ndim, int *size);

template void tridMultiDimBatchPCRCleanMPI<float>(trid_handle<float> &handle, 
                                                  trid_mpi_handle &mpi_handle);

template void tridMultiDimBatchPCRCleanMPI<double>(trid_handle<double> &handle, 
                                                   trid_mpi_handle &mpi_handle);

template void tridMultiDimBatchPCRSolveMPI<float, 0>(trid_handle<float> &handle, 
                                                     trid_mpi_handle &mpi_handle, int solvedim);

template void tridMultiDimBatchPCRSolveMPI<double, 0>(trid_handle<double> &handle, 
                                                     trid_mpi_handle &mpi_handle, int solvedim);

template void tridMultiDimBatchPCRSolveMPI<float, 1>(trid_handle<float> &handle, 
                                                     trid_mpi_handle &mpi_handle, int solvedim);

template void tridMultiDimBatchPCRSolveMPI<double, 1>(trid_handle<double> &handle, 
                                                     trid_mpi_handle &mpi_handle, int solvedim);
