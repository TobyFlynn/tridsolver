#include "hip/hip_runtime.h"
#include "trid_mpi.h"
#include "trid_strided_multidim_pcr_mpi.hpp"

#include <cmath>

template<typename REAL, int INC>
void tridMultiDimBatchPCRSolveMPI(trid_handle<REAL> &handle, trid_mpi_handle &mpi_handle, 
                                  int solvedim) {
  // For now assume 1 MPI proc per GPU
  
  // Allocate aa, cc, dd
  REAL *aa = NULL;
  REAL *cc = NULL;
  REAL *dd = NULL;
  hipMalloc(&aa, sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2]);
  hipMalloc(&cc, sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2]);
  hipMalloc(&dd, sizeof(REAL) * handle.pads[0] * handle.size[1] * handle.size[2]);
  
  // TODO Copy memory from Host to GPU
  
  if(solvedim == 0) {
    // Call forwards pass
    const int numTrids = handle.size[1] * handle.size[2];
    const int length = handle.size[0];
    const int stride = 1;
    const int batchSize = 1;
    const int batchStride = handle.pads[0];
    const int regStoreSize = 8;
    const int threadsPerTrid = (int)ceil((double)length / (double)regStoreSize);
    
    // Work out number of blocks and threads needed
    int totalThreads = threadsPerTrid * numTrids;
    int nThreads = 512;
    int nBlocks = 1;
    if(totalThreads < 512) {
      nThreads = totalThreads;
    } else {
      nBlocks = (int)ceil((double)totalThreads / (double)nThreads);
    }
    
    int reducedSize = threadsPerTrid * 2;
    // TODO change to one interwoven array once algorithm is working
    REAL *aa_r = NULL;
    REAL *cc_r = NULL;
    REAL *dd_r = NULL;
    hipMalloc(&aa_r, sizeof(REAL) * reducedSize * numTrids);
    hipMalloc(&cc_r, sizeof(REAL) * reducedSize * numTrids);
    hipMalloc(&dd_r, sizeof(REAL) * reducedSize * numTrids);
    
    // Call forwards pass
    batched_trid_forwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(handle.a, handle.b, handle.c, 
                                handle.du, aa, cc, dd, aa_r, cc_r, dd_r, length, stride, 
                                numTrids, batchSize, batchStride, threadsPerTrid);
    
    // Call PCR reduced (modified to include MPI comm as reduced system will 
    // be spread over nodes)
    batched_trid_reduced<REAL, regStoreSize>(aa_r, cc_r, dd_r, numTrids, reducedSize, solvedim, threadsPerTrid, 
                               nBlocks, nThreads, length, mpi_handle);
    
    // Call backwards pass
    if(INC) {
      batched_trid_backwardsInc_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.h_u, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
    } else {
      batched_trid_backwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.du, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
    }
    
    // Free memory
    hipFree(aa_r);
    hipFree(cc_r);
    hipFree(dd_r);
  } else if(solvedim == 1) {
    // Call forwards pass
    const int numTrids = handle.size[0] * handle.size[2];
    const int length = handle.size[1];
    const int stride = handle.pads[0];
    const int batchSize = handle.size[0];
    const int batchStride = handle.pads[0] * handle.size[1];
    const int regStoreSize = 8;
    const int threadsPerTrid = (int)ceil((double)length / (double)regStoreSize);
    
    // Work out number of blocks and threads needed
    int totalThreads = threadsPerTrid * numTrids;
    int nThreads = 512;
    int nBlocks = 1;
    if(totalThreads < 512) {
      nThreads = totalThreads;
    } else {
      nBlocks = (int)ceil((double)totalThreads / (double)nThreads);
    }
    
    int reducedSize = threadsPerTrid * 2;
    // TODO change to one interwoven array once algorithm is working
    REAL *aa_r = NULL;
    REAL *cc_r = NULL;
    REAL *dd_r = NULL;
    hipMalloc(&aa_r, sizeof(REAL) * reducedSize * numTrids);
    hipMalloc(&cc_r, sizeof(REAL) * reducedSize * numTrids);
    hipMalloc(&dd_r, sizeof(REAL) * reducedSize * numTrids);
    
    // Call forwards pass
    batched_trid_forwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(handle.a, handle.b, handle.c, 
                                handle.du, aa, cc, dd, aa_r, cc_r, dd_r, length, stride, 
                                numTrids, batchSize, batchStride, threadsPerTrid);
    
    // Call PCR reduced (modified to include MPI comm as reduced system will 
    // be spread over nodes)
    batched_trid_reduced<REAL, regStoreSize>(aa_r, cc_r, dd_r, numTrids, reducedSize, solvedim, threadsPerTrid, 
                               nBlocks, nThreads, length, mpi_handle);
    
    // Call backwards pass
    if(INC) {
      batched_trid_backwardsInc_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.h_u, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
    } else {
      batched_trid_backwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.du, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
    }
    
    // Free memory
    hipFree(aa_r);
    hipFree(cc_r);
    hipFree(dd_r);
  } else if(solvedim == 2) {
    // Call forwards pass
    const int numTrids = handle.size[0] * handle.size[1];
    const int length = handle.size[2];
    const int stride = handle.pads[0] * handle.size[1];
    const int batchSize = handle.size[0];
    const int batchStride = handle.pads[0];
    const int regStoreSize = 8;
    const int threadsPerTrid = (int)ceil((double)length / (double)regStoreSize);
    
    // Work out number of blocks and threads needed
    int totalThreads = threadsPerTrid * numTrids;
    int nThreads = 512;
    int nBlocks = 1;
    if(totalThreads < 512) {
      nThreads = totalThreads;
    } else {
      nBlocks = (int)ceil((double)totalThreads / (double)nThreads);
    }
    
    int reducedSize = threadsPerTrid * 2;
    // TODO change to one interwoven array once algorithm is working
    REAL *aa_r = NULL;
    REAL *cc_r = NULL;
    REAL *dd_r = NULL;
    hipMalloc(&aa_r, sizeof(REAL) * reducedSize * numTrids);
    hipMalloc(&cc_r, sizeof(REAL) * reducedSize * numTrids);
    hipMalloc(&dd_r, sizeof(REAL) * reducedSize * numTrids);
    
    // Call forwards pass
    batched_trid_forwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(handle.a, handle.b, handle.c, 
                                handle.du, aa, cc, dd, aa_r, cc_r, dd_r, length, stride, 
                                numTrids, batchSize, batchStride, threadsPerTrid);
    
    // Call PCR reduced (modified to include MPI comm as reduced system will 
    // be spread over nodes)
    batched_trid_reduced<REAL, regStoreSize>(aa_r, cc_r, dd_r, numTrids, reducedSize, solvedim, threadsPerTrid, 
                               nBlocks, nThreads, length, mpi_handle);
    
    // Call backwards pass
    if(INC) {
      batched_trid_backwardsInc_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.h_u, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
    } else {
      batched_trid_backwards_kernel<REAL, regStoreSize><<<nBlocks, nThreads>>>(aa, cc, dd, dd_r, handle.du, 
                                                      length, stride, numTrids, batchSize, 
                                                      batchStride, threadsPerTrid);
    }
    
    // Free memory
    hipFree(aa_r);
    hipFree(cc_r);
    hipFree(dd_r);
  }
}

template void tridMultiDimBatchPCRSolveMPI<float, 0>(trid_handle<float> &handle, 
                                                     trid_mpi_handle &mpi_handle, int solvedim);

template void tridMultiDimBatchPCRSolveMPI<double, 0>(trid_handle<double> &handle, 
                                                     trid_mpi_handle &mpi_handle, int solvedim);

template void tridMultiDimBatchPCRSolveMPI<float, 1>(trid_handle<float> &handle, 
                                                     trid_mpi_handle &mpi_handle, int solvedim);

template void tridMultiDimBatchPCRSolveMPI<double, 1>(trid_handle<double> &handle, 
                                                     trid_mpi_handle &mpi_handle, int solvedim);
